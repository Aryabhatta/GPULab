#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: convolution
* file:    convolution_gpu.cu
*
* 
\******* PLEASE ENTER YOUR CORRECT STUDENT LOGIN, NAME AND ID BELOW *********/
const char* gpu_studentLogin = "p110";
const char* gpu_studentName  = "Shrikant Vinchurkar";
const int   gpu_studentID    = 03636145;
/****************************************************************************\
*
* In this file the following methods have to be edited or completed:
*
* gpu_convolutionGrayImage_gm_d
* gpu_convolutionGrayImage_gm_cm_d
* gpu_convolutionGrayImage_sm_d
* gpu_convolutionGrayImage_sm_cm_d
* gpu_convolutionGrayImage_dsm_cm_d
* gpu_convolutionInterleavedRGB_dsm_cm_d
* gpu_convolutionInterleavedRGB_tex_cm_d
*
\****************************************************************************/


#include "convolution_gpu.cuh"

#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <iostream>


#define TEXTURE_OFFSET      0.5f  // offset for indexing textures

#define BW                  16    // block width
#define BH                  16    // block height
#define MAXKERNELRADIUS     20    // maximum allowed kernel radius

#define MAXKERNELSIZE       ( 1+2*MAXKERNELRADIUS)*( 1+2*MAXKERNELRADIUS)
#define MAXSHAREDMEMSIZE    (BW+2*MAXKERNELRADIUS)*(BH+2*MAXKERNELRADIUS)

#if (MAXSHAREDMEMSIZE > 4000)   // Note: MAXSHAREDMEMSIZE <= 4000 should hold for most graphic cards to work
  #error "This program will most likely not run properly because of insufficient shared memory, please reduce BW/BH/MAXKERNELRADIUS!"
#endif



// constant memory block on device
__constant__ float constKernel[MAXKERNELSIZE];

// texture memory and descriptor
hipChannelFormatDesc tex_Image_desc = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_Image;

hipChannelFormatDesc tex_Image_descF4 = hipCreateChannelDesc<float4>();
texture<float4, 2, hipReadModeElementType> tex_ImageF4;


const char* gpu_getStudentLogin() { return gpu_studentLogin; };
const char* gpu_getStudentName()  { return gpu_studentName; };
int         gpu_getStudentID()    { return gpu_studentID; };
bool gpu_checkStudentData() { return strcmp(gpu_studentLogin, "p010") != 0 && strcmp(gpu_studentName, "John Doe") != 0 && gpu_studentID != 1234567; };
bool gpu_checkStudentNameAndID() { return strcmp(gpu_studentName, "John Doe") != 0 && gpu_studentID != 1234567; };



//----------------------------------------------------------------------------
// Gray Image Functions
//----------------------------------------------------------------------------


// mode 1 (gray): using global memory only
__global__ void gpu_convolutionGrayImage_gm_d(const float *inputImage, const float *kernel, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch, size_t kPitch)
{ 
  // ### implement me ###
  int k = 0, l = 0;
  
  const int kWidth  = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;
  
  int ix = threadIdx.x + blockIdx.x * blockDim.x;
  int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
  // initialising output image
  if( ix < iWidth && iy < iHeight )
    outputImage[ iy * iPitch + ix] = 0.0;
  
  int IndX = 0, IndY = 0;
  int shiftX = 0, shiftY = 0;
  
  shiftX = ix - kRadiusX;
  shiftY = iy - kRadiusY;

  // ### implement a convolution ### 
  for( k = 0; k < kWidth; k++ )
  {
    for( l = 0; l < kHeight; l++)
  	{	
  		IndX = shiftX + k;
  		IndY = shiftY + l;

  		// handling boundary conditions
  		if( IndX < 0 ){			IndX = 0;  		}
  		if( IndY < 0){ 			IndY = 0;  		}
  		
  		if( IndX > iWidth -1){		IndX = iWidth - 1;  		}
  		if( IndY > iHeight -1){		IndY = iHeight- 1;	 		}

  		if( ix < iWidth && iy < iHeight ) // guards
  		  outputImage[ iy * iPitch + ix ] += inputImage[ IndY * iPitch + IndX ] * \
  			   		                         kernel[ l * kPitch + k];
  	  }
  	} //replacing kWidth by kPitch gives correct indexing in kernel
  	  // same goes as iPitch
}



// mode 2 (gray): using global memory and constant memory for kernel
__global__ void gpu_convolutionGrayImage_gm_cm_d(const float *inputImage, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch)
{  
	  // ### implement me ###
	  int k = 0, l = 0;
	  
	  const int kWidth  = (kRadiusX << 1) + 1;
	  const int kHeight = (kRadiusY << 1) + 1;
	  
	  int ix = threadIdx.x + blockIdx.x * blockDim.x;
	  int iy = threadIdx.y + blockIdx.y * blockDim.y;
	    
	  // initialising output image
	  if( ix < iWidth && iy < iHeight )
	    outputImage[ iy * iPitch + ix] = 0.0;
	  
	  int IndX = 0, IndY = 0;
	  int shiftX = 0, shiftY = 0;
	    
	  shiftX = ix - kRadiusX;
	  shiftY = iy - kRadiusY;

	  // ### implement a convolution ### 
	  for( k = 0; k < kWidth; k++ )
	  {
	    for( l = 0; l < kHeight; l++)
	  	{	
	  		IndX = shiftX + k;
	  		IndY = shiftY + l;

	  		// handling boundary conditions
	  		if( IndX < 0 ){			IndX = 0;	  		}
	  		if( IndY < 0) {			IndY = 0;	  		}
	  		
	  		if( IndX > iWidth -1){		IndX = iWidth - 1;	  		}
	  		if( IndY > iHeight -1){		IndY = iHeight- 1;	  		}

	  		if( ix < iWidth && iy < iHeight ) // guards
	  		  outputImage[ iy * iPitch + ix ] += inputImage[ IndY * iPitch + IndX ] * \
	  					                         constKernel[ l * kWidth + k];
	  	  }
	  	} // need nt replace kWidth by kPitch here   
}


// mode 3 (gray): using shared memory for image and globel memory for kernel access
__global__ void gpu_convolutionGrayImage_sm_d(const float *inputImage, const float *kernel, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch, size_t kPitch)
{
  // make use of the constant MAXSHAREDMEMSIZE in order to define the shared memory size

  // ### implement me ###	
  __shared__ float input[ MAXSHAREDMEMSIZE ];
  
  const int kWidth  = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;
    
  // Copy input array equal to blocksize + 2 * kRadius  
  // size of block to be copied = (BW + 2 * kRadiusX)  * (BH + 2* kRadiusY)
  int sWidth  = blockDim.x + 2* kRadiusX;
  int sHeight = blockDim.y + 2* kRadiusY;
    
  int InputSize = sWidth * sHeight;
  int blockSize = blockDim.x * blockDim.y;

  int NoLoops = 0;
  if( InputSize % blockSize == 0)
  {
    NoLoops = InputSize / blockSize;
  }
  else
  {
    NoLoops = (InputSize/blockSize) + 1;
  }
  
  int ix = threadIdx.x + blockIdx.x * blockDim.x; // global indexes for img (input & output) 
  int iy = threadIdx.y + blockIdx.y * blockDim.y;
  
  // for example 16*16 blocksize & 16*16 no of blocks,ix & iy value from 0...255
  // here,though blockSize=16*16, #blocks_inX = iWidth/16,#blocks_inY=iHeight/16
   
  // shifting global indices to align it to required start point
  int start_X = (blockIdx.x * blockDim.x - kRadiusX);
  int start_Y = (blockIdx.y * blockDim.y - kRadiusY);
  
  int IndX = 0;
  int IndY = 0;

  // aligning locations equal to blockSize in input (without offset)
  int inpStart = threadIdx.x + threadIdx.y * blockDim.x;
  int inputIdx = 0;
  int inputOffset = 0; // initial offset

  for( int i = 0; i < NoLoops ; i++ )
  {    
    if( inputIdx < InputSize ) // guard
    {
    	inputOffset = i * blockSize; // 0 offset in first loop, +blockSize in next each
   	    inputIdx = inpStart + inputOffset; // final index for input

   	    // aligning global index as per read sequence in Input
        IndX = start_X  + inputIdx % sWidth;
        IndY = start_Y  + inputIdx / sHeight;

	    // guards
        if ( IndX < 0 ) { IndX  = 0; }
        if ( IndY < 0 ) { IndY  = 0; }

        if ( IndX > iWidth -1 ) { IndX  = iWidth -1; }
        if ( IndY > iHeight-1 ) { IndY  = iHeight-1; }

        input[ inputIdx ] = inputImage[ IndY * iPitch + IndX ];
    }
  }
  
  // syncthreads
  __syncthreads();

  // initialising output image
  if( ix < iWidth && iy < iHeight) //guards
    outputImage[ iy * iPitch + ix ] = 0.0; // indexing globally

  int tx = threadIdx.x;
  int ty = threadIdx.y;
   
  int index = 0, k = 0, l = 0;
    
 // ### implement a convolution ###
  for( k = 0; k < kWidth; k++ )
  {
	for( l = 0; l < kHeight; l++)
	{
		// index for input array
	   index = (ty+l) * sWidth + (tx+k);
	   
	   if( ix < iWidth &&  iy < iHeight) //guards
	     if( index < InputSize )
	       outputImage[ iy * iPitch + ix ] += input[ index ] * \
		   								      kernel[ l * kPitch + k];
	}
  }  
}


// mode 4 (gray): using shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_sm_cm_d(const float *inputImage, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch)
{
  // make use of the constant MAXSHAREDMEMSIZE in order to define the shared memory size

  // ### implement me ###
	__shared__ float input[ MAXSHAREDMEMSIZE ];
	  
	  const int kWidth  = (kRadiusX << 1) + 1;
	  const int kHeight = (kRadiusY << 1) + 1;
	    
	  // Copy input array equal to blocksize + 2 * kRadius  
	  // size of block to be copied = (BW + 2 * kRadiusX)  * (BH + 2* kRadiusY)
	  int sWidth  = blockDim.x + 2* kRadiusX;
	  int sHeight = blockDim.y + 2* kRadiusY;
	    
	  int InputSize = sWidth * sHeight;
	  int blockSize = blockDim.x * blockDim.y;

	  int NoLoops = 0;
	  if( InputSize % blockSize == 0)
	  {
	    NoLoops = InputSize / blockSize;
	  }
	  else
	  {
	    NoLoops = (InputSize/blockSize) + 1;
	  }
	  
	  int ix = threadIdx.x + blockIdx.x * blockDim.x; // global indexes for img (input & output) 
	  int iy = threadIdx.y + blockIdx.y * blockDim.y;
	  
	  // for example 16*16 blocksize & 16*16 no of blocks,ix & iy value from 0...255
	  // here,though blockSize=16*16, #blocks_inX = iWidth/16,#blocks_inY=iHeight/16
	   
	  // shifting global indices to align it to requires start point
	  int start_X = (blockIdx.x * blockDim.x - kRadiusX);
	  int start_Y = (blockIdx.y * blockDim.y - kRadiusY);
	  
	  int IndX = 0;
	  int IndY = 0;

	  // aligning locations equal to blockSize in input (without offset)
	  int inpStart = threadIdx.x + threadIdx.y * blockDim.x;
	  int inputIdx = 0;
	  int inputOffset = 0; // initial offset

	  for( int i = 0; i < NoLoops ; i++ )
	  {    
	    if( inputIdx < InputSize ) // guard
	    {
	    	inputOffset = i * blockSize; // 0 in first loop, +blockSize in next each
	   	    inputIdx = inpStart + inputOffset; // final index for input

	   	    // aligning global index as per read sequence in Input
	        IndX = start_X  + inputIdx % sWidth;
	        IndY = start_Y  + inputIdx / sHeight;

		    // guards
	        if ( IndX < 0 ) { IndX  = 0; }
	        if ( IndY < 0 ) { IndY  = 0; }

	        if ( IndX > iWidth -1 ) { IndX  = iWidth -1; }
	        if ( IndY > iHeight-1 ) { IndY  = iHeight-1; }

	        input[ inputIdx ] = inputImage[ IndY * iPitch + IndX ];
	    }
	  }
	  
	  // syncthreads
	  __syncthreads();

	  // initialising output image
	  if( ix < iWidth && iy < iHeight) //guards
	    outputImage[ iy * iPitch + ix ] = 0.0; // indexing globally

	  // anchoring points in array input
	  int tx = threadIdx.x;
	  int ty = threadIdx.y;
	   
	  int index = 0, k = 0, l = 0;
	    
	 // ### implement a convolution ###
	  for( k = 0; k < kWidth; k++ )
	  {
		for( l = 0; l < kHeight; l++)
		{
			// index for input array
		   index = (ty+l) * sWidth + (tx+k);
		   
		   if( ix < iWidth &&  iy < iHeight) //guards
		     if( index < InputSize )
		       outputImage[ iy * iPitch + ix ] += input[ index ] * \
			   								      constKernel[ l * kWidth + k];
		}
	  }  
} 


// mode 5 (gray): using dynamically allocated shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_dsm_cm_d(const float *inputImage, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch)
{

  // ### implement me ###  
	extern __shared__ float input[];
	  
	const int kWidth  = (kRadiusX << 1) + 1;
	const int kHeight = (kRadiusY << 1) + 1;
	    
	  // Copy input array equal to blocksize + 2 * kRadius  
	  // size of block to be copied = (BW + 2 * kRadiusX)  * (BH + 2* kRadiusY)
	  int sWidth  = blockDim.x + 2* kRadiusX;
	  int sHeight = blockDim.y + 2* kRadiusY;
	    
	  int InputSize = sWidth * sHeight;
	  int blockSize = blockDim.x * blockDim.y;

	  int NoLoops = 0;
	  if( InputSize % blockSize == 0)
	  {
	    NoLoops = InputSize / blockSize;
	  }
	  else
	  {
	    NoLoops = (InputSize/blockSize) + 1;
	  }
	  
	  int ix = threadIdx.x + blockIdx.x * blockDim.x; // global indexes for img (input & output) 
	  int iy = threadIdx.y + blockIdx.y * blockDim.y;
	  
	  // for example 16*16 blocksize & 16*16 no of blocks,ix & iy value from 0...255
	  // here,though blockSize=16*16, #blocks_inX = iWidth/16,#blocks_inY=iHeight/16
	   
	  // shifting global indices to align it to requires start point
	  int start_X = (blockIdx.x * blockDim.x - kRadiusX);
	  int start_Y = (blockIdx.y * blockDim.y - kRadiusY);
	  
	  int IndX = 0;
	  int IndY = 0;

	  // aligning locations equal to blockSize in input (without offset)
	  int inpStart = threadIdx.x + threadIdx.y * blockDim.x;
	  int inputIdx = 0;
	  int inputOffset = 0; // initial offset

	  for( int i = 0; i < NoLoops ; i++ )
	  {    
	    if( inputIdx < InputSize ) // guard
	    {
	    	inputOffset = i * blockSize; // 0 in first loop, +blockSize in next each
	   	    inputIdx = inpStart + inputOffset; // final index for input

	   	    // aligning global index as per read sequence in Input
	        IndX = start_X  + inputIdx % sWidth;
	        IndY = start_Y  + inputIdx / sHeight;

		    // guards
	        if ( IndX < 0 ) { IndX  = 0; }
	        if ( IndY < 0 ) { IndY  = 0; }

	        if ( IndX > iWidth -1 ) { IndX  = iWidth -1; }
	        if ( IndY > iHeight-1 ) { IndY  = iHeight-1; }

	        input[ inputIdx ] = inputImage[ IndY * iPitch + IndX ];
	    }
	  }
	  
	  // syncthreads
	  __syncthreads();

	  // initialising output image
	  if( ix < iWidth && iy < iHeight) //guards
	    outputImage[ iy * iPitch + ix ] = 0.0f; // indexing globally

	  // anchoring points in array input
	  int tx = threadIdx.x;
	  int ty = threadIdx.y;
	   
	  int index = 0, k = 0, l = 0;
	    
	 // ### implement a convolution ###
	  for( k = 0; k < kWidth; k++ )
	  {
		for( l = 0; l < kHeight; l++)
		{
			// index for input array
		   index = (ty+l) * sWidth + (tx+k);
		   
		   if( ix < iWidth &&  iy < iHeight) //guards
		     if( index < InputSize )
		       outputImage[ iy * iPitch + ix ] += input[ index ] * \
			   								      constKernel[ l * kWidth + k];
		}
	  }
} 




// mode 6 (gray): using texture memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_tex_cm_d(float *outputImage,
    int iWidth, int iHeight, int kRadiusX, int kRadiusY,
    size_t iPitch)
{
	// updated code from Martin's last mail
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= iWidth || y >= iHeight) return;

	const float xx = (float)(x) + TEXTURE_OFFSET;
	const float yy = (float)(y) + TEXTURE_OFFSET;
	const int kWidth  = (kRadiusX<<1) + 1;

	float value = 0.0f;

	for (int yk = -kRadiusY; yk <= kRadiusY; yk++)
	  for (int xk = -kRadiusX; xk <= kRadiusX; xk++)
	    value += tex2D(tex_Image, xx-xk, yy-yk) * constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX];

	outputImage[y*iPitch + x] = value;
}



void gpu_convolutionGrayImage(const float *inputImage, const float *kernel, float *outputImage, 
                              int iWidth, int iHeight, int kRadiusX, int kRadiusY, int mode)
{
  size_t iPitchBytes, kPitchBytes;
  size_t iPitch, kPitch;
  float *d_inputImage;
  float *d_kernel;
  float *d_outputImage;

  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  assert(kWidth*kHeight <= MAXKERNELSIZE);
  
  // allocate device memory
  cutilSafeCall( hipMallocPitch( (void**)&d_inputImage, &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&d_outputImage, &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&d_kernel, &kPitchBytes, kWidth*sizeof(float), kHeight ) );  
  
  iPitch = iPitchBytes/sizeof(float);
  kPitch = kPitchBytes/sizeof(float);
  
  //std::cout << "iPitchBytes=" << iPitchBytes << " iPitch=" << iPitch << " kPitchBytes=" << kPitchBytes << " kPitch=" << kPitch << std::endl;  
  
  cutilSafeCall( hipMemcpy2D(d_inputImage, iPitchBytes, inputImage, iWidth*sizeof(float), iWidth*sizeof(float), iHeight, hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy2D(d_kernel, kPitchBytes, kernel, kWidth*sizeof(float), kWidth*sizeof(float), kHeight, hipMemcpyHostToDevice) );

  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  gpu_bindTextureMemory(d_inputImage, iWidth, iHeight, iPitchBytes);

  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  /*std::cout << "ImageWidth=" << iWidth << " ImageHeight=" << iHeight << std::endl;
  std::cout << "blockWidth=" << BW << " blockHeight=" << BH << std::endl;
  std::cout << "gridSize.x=" << gridSize.x << " gridSize.y=" << gridSize.y << std::endl;*/ 

  // invoke the kernel of your choice here
  const int smSize =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float);  

  switch(mode) {
    case 1:
      gpu_convolutionGrayImage_gm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
      break;
    case 2:
      gpu_convolutionGrayImage_gm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    case 3:
      gpu_convolutionGrayImage_sm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
      break;
    case 4:
      gpu_convolutionGrayImage_sm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    case 5:
      gpu_convolutionGrayImage_dsm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    case 6:
      gpu_convolutionGrayImage_tex_cm_d<<<gridSize,blockSize>>>(d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    default:
      std::cout << "gpu_convolutionGrayImage() Warning: mode " << mode << " is not supported." << std::endl;
  }

  cutilSafeCall( hipDeviceSynchronize() );
  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float), d_outputImage, iPitchBytes, iWidth*sizeof(float), iHeight, hipMemcpyDeviceToHost) );
  

  // free memory
  gpu_unbindTextureMemory();
  cutilSafeCall( hipFree(d_inputImage) );
  cutilSafeCall( hipFree(d_outputImage) );
  cutilSafeCall( hipFree(d_kernel) );
}




//----------------------------------------------------------------------------
// RGB Image Functions (for separated color channels)
//----------------------------------------------------------------------------



void gpu_convolutionRGB(const float *inputImage, const float *kernel, float *outputImage, 
                        int iWidth, int iHeight, int kRadiusX, int kRadiusY, int mode)
{
  const int imgSize = iWidth*iHeight;
  gpu_convolutionGrayImage(inputImage, kernel, outputImage, iWidth, iHeight, kRadiusX, kRadiusY, mode);
  gpu_convolutionGrayImage(inputImage+imgSize, kernel, outputImage+imgSize, iWidth, iHeight, kRadiusX, kRadiusY, mode);
  gpu_convolutionGrayImage(inputImage+(imgSize<<1), kernel, outputImage+(imgSize<<1), iWidth, iHeight, kRadiusX, kRadiusY, mode);
}


//----------------------------------------------------------------------------
// RGB Image Functions (for interleaved color channels)
//----------------------------------------------------------------------------


// mode 5 (interleaved): using dynamically allocated shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionInterleavedRGB_dsm_cm_d(const float3 *inputImage, float3 *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitchBytes)
{
  extern __shared__ float3 inputF3[];
	
  const int x = blockIdx.x * blockDim.x + threadIdx.x;// global indexes for img (input & output)
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  float3 value = make_float3(0.0f, 0.0f, 0.0f);

  // ### implement me ###	  
  const int kWidth  = (kRadiusX << 1) + 1;
  const int kHeight = (kRadiusY << 1) + 1;
	    
  // Copy input array equal to blocksize + 2 * kRadius  
  // size of block to be copied = (BW + 2 * kRadiusX)  * (BH + 2* kRadiusY)
  int sWidth  = blockDim.x + 2 * kRadiusX;
  int sHeight = blockDim.y + 2 * kRadiusY;
	    
  int InputSize = sWidth * sHeight;
  int blockSize = blockDim.x * blockDim.y;

  int NoLoops = 0;
  if( InputSize % blockSize == 0)
  {
    NoLoops = InputSize / blockSize;
  }
  else
  {
	NoLoops = (InputSize/blockSize) + 1;
  }
	   
	// shifting global indices to align it to requires start point
	int start_X = (blockIdx.x * blockDim.x - kRadiusX);
	int start_Y = (blockIdx.y * blockDim.y - kRadiusY);
	  
	int IndX = 0;
	int IndY = 0;

	// aligning locations equal to blockSize in input (without offset)
	int inpStart = threadIdx.x + threadIdx.y * blockDim.x;
	int inputIdx = 0;
	int inputOffset = 0; // initial offset

	for( int i = 0; i < NoLoops ; i++ )
	{    
		if( inputIdx < InputSize ) // guard
	    {
	    	inputOffset = i * blockSize; // 0 in first loop, +blockSize in next each
	   	    inputIdx = inpStart + inputOffset; // final index for input

	   	    // aligning global index as per read sequence in Input
	        IndX = start_X  + inputIdx % sWidth;
	        IndY = start_Y  + inputIdx / sHeight;

		    // guards
	        if ( IndX < 0 ) { IndX  = 0; }
	        if ( IndY < 0 ) { IndY  = 0; }

	        if ( IndX > iWidth -1 ) { IndX  = iWidth -1; }
	        if ( IndY > iHeight-1 ) { IndY  = iHeight-1; }

	        inputF3[ inputIdx ] = *((float3*)(((char*)inputImage) + IndY*iPitchBytes) + IndX); 
	    }
	}
	  
	  // syncthreads
	  __syncthreads();

	  // initialising output image
	  if( x < iWidth && y < iHeight) //guards
		  *((float3*)(((char*)outputImage) + y*iPitchBytes) + x) = value; // indexing globally

	  // anchoring points in array input
	  int tx = threadIdx.x;
	  int ty = threadIdx.y;
	   
	  int index = 0, k = 0, l = 0;
	    
	 // ### implement a convolution ###
	  for( k = 0; k < kWidth; k++ )
	  {
		for( l = 0; l < kHeight; l++)
		{
			// index for input array
		   index = (ty+l) * sWidth + (tx+k);
		   	   
		     if( index < InputSize )
		     {
		    	 value.x += inputF3[ index ].x * constKernel[ l * kWidth + k];
		    	 value.y += inputF3[ index ].y * constKernel[ l * kWidth + k];
		    	 value.z += inputF3[ index ].z * constKernel[ l * kWidth + k];
		     }
		}
	  }
	  if( x < iWidth &&  y < iHeight) //guards
	  {
		  *((float3*)(((char*)outputImage) + y*iPitchBytes) + x) = value;
	  }
} 





__global__ void gpu_ImageFloat3ToFloat4_d(const float3 *inputImage, float4 *outputImage, int iWidth, int iHeight, size_t iPitchBytes, size_t oPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= iWidth || y >= iHeight) return;

  float3 rgb = *((float3*)((char*)inputImage + y*iPitchBytes)+x);
  *((float4*)(((char*)outputImage) + y*oPitchBytes)+ x) = make_float4(rgb.x, rgb.y, rgb.z, 0.0f);
}




// mode 6 (interleaved): using texture memory for image and constant memory for kernel access
__global__ void gpu_convolutionInterleavedRGB_tex_cm_d(float3 *outputImage,
    int iWidth, int iHeight, int kRadiusX, int kRadiusY, size_t oPitchBytes)
{

  // ### implement me ### 
	// updated code from Martin's last mail
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= iWidth || y >= iHeight) return;

	const float xx = (float)(x) + TEXTURE_OFFSET;
	const float yy = (float)(y) + TEXTURE_OFFSET;
	const int kWidth  = (kRadiusX<<1) + 1;

	float4 value = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float3 imgValue = make_float3(0.0f, 0.0f, 0.0f);
	
	for (int yk = -kRadiusY; yk <= kRadiusY; yk++)
	  for (int xk = -kRadiusX; xk <= kRadiusX; xk++)
	  {
		value = tex2D(tex_ImageF4, xx-xk, yy-yk);
		imgValue.x += value.x * constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX];
	    imgValue.y += value.y * constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX];
	    imgValue.z += value.z * constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX];
	  }
	
	*((float3*)(((char*)outputImage) + y*oPitchBytes) + x) = imgValue;
}



void gpu_convolutionInterleavedRGB(const float *inputImage, const float *kernel, float *outputImage,
                                   int iWidth, int iHeight, int kRadiusX, int kRadiusY, int mode)
{
  size_t iPitchBytesF3, iPitchBytesF4;
  float3 *d_inputImageF3, *d_outputImageF3;
  float4 *d_inputImageF4;
  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  //  allocate memory and copy data
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_outputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF4), &iPitchBytesF4, iWidth*sizeof(float4), iHeight ) );

  cutilSafeCall( hipMemcpy2D(d_inputImageF3, iPitchBytesF3, inputImage, iWidth*sizeof(float3), iWidth*sizeof(float3), iHeight, hipMemcpyHostToDevice) );

  dim3 blockSize(BW,BH);
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  int smSizeF3 =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float3);
  
  // convert image from float3* to float4*
  gpu_ImageFloat3ToFloat4_d<<<gridSize, blockSize>>>(d_inputImageF3, d_inputImageF4, iWidth, iHeight, iPitchBytesF3, iPitchBytesF4);
  
  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  gpu_bindTextureMemoryF4(d_inputImageF4, iWidth, iHeight, iPitchBytesF4);


  switch(mode) {
    case 1:
    case 2:
    case 3:
    case 4:
      std::cout << "gpu_convolutionInterleavedRGB() Warning: mode " << mode << " is not supported." << std::endl;
      break;
    case 5:
      gpu_convolutionInterleavedRGB_dsm_cm_d<<<gridSize,blockSize,smSizeF3>>>(d_inputImageF3, d_outputImageF3,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
      break;
    case 6:
      gpu_convolutionInterleavedRGB_tex_cm_d<<<gridSize,blockSize>>>(d_outputImageF3,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
      break;
    default:
      std::cout << "gpu_convolutionInterleavedRGB() Warning: mode " << mode << " is not supported." << std::endl;
  }

  cutilSafeCall( hipDeviceSynchronize() );
  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float3), d_outputImageF3, iPitchBytesF3, iWidth*sizeof(float3), iHeight, hipMemcpyDeviceToHost) );


  // free memory
  gpu_unbindTextureMemoryF4();
  cutilSafeCall( hipFree(d_inputImageF4) );
  cutilSafeCall( hipFree(d_inputImageF3) );
  cutilSafeCall( hipFree(d_outputImageF3) );
}












//----------------------------------------------------------------------------
// Benchmark Functions
//----------------------------------------------------------------------------




void gpu_convolutionKernelBenchmarkGrayImage(const float *inputImage, const float *kernel, float *outputImage,
                                             int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                             int numKernelTestCalls)
{
  size_t iPitchBytes, kPitchBytes;
  size_t iPitch, kPitch;
  clock_t startTime, endTime;
  float *d_inputImage, *d_kernel, *d_outputImage;
  float fps;

  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  assert(kWidth*kHeight <= MAXKERNELSIZE);

  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  int smSize =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float);

  //  allocate memory and copy data
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImage), &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_outputImage), &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_kernel), &kPitchBytes, kWidth*sizeof(float), kHeight ) );   
  iPitch = iPitchBytes/sizeof(float);
  kPitch = kPitchBytes/sizeof(float);
  
  cutilSafeCall( hipMemcpy2D(d_inputImage, iPitchBytes, inputImage, iWidth*sizeof(float), iWidth*sizeof(float), iHeight, hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy2D(d_kernel, kPitchBytes, kernel, kWidth*sizeof(float), kWidth*sizeof(float), kHeight, hipMemcpyHostToDevice) );

  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  gpu_bindTextureMemory(d_inputImage, iWidth, iHeight, iPitchBytes);

  // --- global memory only ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_gm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  std::cout << fps << " fps - global memory only\n";
  
  
  // --- global memory for image and constant memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_gm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  std::cout << fps << " fps - global memory for image & constant memory for kernel access\n";


  // --- shared memory for image and global memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_sm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  std::cout << fps << " fps - shared memory for image & global memory for kernel access\n";

  
  // --- shared memory for image and constant memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_sm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  std::cout << fps << " fps - shared memory for image & constant memory for kernel access\n";


   // --- shared memory for image and constant memory for kernel access ---  
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_dsm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  std::cout << fps << " fps - dyn. shared memory for image & const memory for kernel access\n";



  // --- texture memory for image and constant memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_tex_cm_d<<<gridSize,blockSize>>>(d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  std::cout << fps << " fps - texture memory for image & const memory for kernel access\n";


  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float), d_outputImage, iPitchBytes, iWidth*sizeof(float), iHeight, hipMemcpyDeviceToHost) );

  // free memory
  gpu_unbindTextureMemory();
  cutilSafeCall( hipFree(d_inputImage) );
  cutilSafeCall( hipFree(d_outputImage) );
  cutilSafeCall( hipFree(d_kernel) );
}



void gpu_convolutionKernelBenchmarkInterleavedRGB(const float *inputImage, const float *kernel, float *outputImage,
                                                  int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                                  int numKernelTestCalls)
{
  size_t iPitchBytesF3, iPitchBytesF4;
  clock_t startTime, endTime;
  float3 *d_inputImageF3, *d_outputImageF3;
  float4 *d_inputImageF4;
  float fps;

  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  assert(kWidth*kHeight <= MAXKERNELSIZE);

  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  int smSizeF3 =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float3);

  //  allocate memory and copy data
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF4), &iPitchBytesF4, iWidth*sizeof(float4), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_outputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );   
  cutilSafeCall( hipMemcpy2D(d_inputImageF3, iPitchBytesF3, inputImage, iWidth*sizeof(float3), iWidth*sizeof(float3), iHeight, hipMemcpyHostToDevice) );

  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  
  
  // --- shared memory for interleaved image and constant memory for kernel access ---  
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionInterleavedRGB_dsm_cm_d<<<gridSize,blockSize,smSizeF3>>>(d_inputImageF3, d_outputImageF3, iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC * 3;
  std::cout << fps << " fps - dyn. shared mem for interleaved img & const mem for kernel\n";


  // --- texture memory for interleaved image and constant memory for kernel access ---
  gpu_ImageFloat3ToFloat4_d<<<gridSize, blockSize>>>(d_inputImageF3, d_inputImageF4, iWidth, iHeight, iPitchBytesF3, iPitchBytesF4);
  gpu_bindTextureMemoryF4(d_inputImageF4, iWidth, iHeight, iPitchBytesF4);

  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionInterleavedRGB_tex_cm_d<<<gridSize,blockSize>>>(d_outputImageF3, iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC * 3;
  std::cout << fps << " fps - texture mem for interleaved img & const mem for kernel access\n";


  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float3), d_outputImageF3, iPitchBytesF3, iWidth*sizeof(float3), iHeight, hipMemcpyDeviceToHost) );
  

  // free memory
  gpu_unbindTextureMemoryF4();
  cutilSafeCall( hipFree(d_inputImageF3) );
  cutilSafeCall( hipFree(d_outputImageF3) );
  cutilSafeCall( hipFree(d_inputImageF4) );
}





void gpu_bindConstantMemory(const float *kernel, int size) 
{
  cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(constKernel), kernel, size*sizeof(float)) );
}



void gpu_bindTextureMemory(float *d_inputImage, int iWidth, int iHeight, size_t iPitchBytes)
{
  // >>>> prepare usage of texture memory
  tex_Image.addressMode[0] = hipAddressModeClamp;
  tex_Image.addressMode[1] = hipAddressModeClamp;
  tex_Image.filterMode = hipFilterModeLinear;
  tex_Image.normalized = false;
  // <<<< prepare usage of texture memory

  cutilSafeCall( hipBindTexture2D(0, &tex_Image, d_inputImage, &tex_Image_desc, iWidth, iHeight, iPitchBytes) );
}


void gpu_unbindTextureMemory()
{
  cutilSafeCall( hipUnbindTexture(tex_Image) );
}



void gpu_bindTextureMemoryF4(float4 *d_inputImageF4, int iWidth, int iHeight, size_t iPitchBytesF4)
{
  // >>>> prepare usage of texture memory
  tex_ImageF4.addressMode[0] = hipAddressModeClamp;
  tex_ImageF4.addressMode[1] = hipAddressModeClamp;
  tex_ImageF4.filterMode = hipFilterModeLinear;
  tex_ImageF4.normalized = false;
  // <<<< prepare usage of texture memory

  cutilSafeCall( hipBindTexture2D(0, &tex_ImageF4, d_inputImageF4, &tex_Image_descF4, iWidth, iHeight, iPitchBytesF4) );
}


void gpu_unbindTextureMemoryF4()
{
  cutilSafeCall( hipUnbindTexture(tex_ImageF4) );
}

